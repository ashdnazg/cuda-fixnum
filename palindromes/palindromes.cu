#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <cassert>
#include <cinttypes>

#include "fixnum/warp_fixnum.cu"
#include "array/fixnum_array.h"
#include "functions/modexp.cu"
#include "functions/multi_modexp.cu"
#include "modnum/modnum_monty_redc.cu"
#include "modnum/modnum_monty_cios.cu"

#include "uint256_t.h"

using namespace std;
using namespace cuFIXNUM;

template< typename fixnum >
struct add {
    __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
        fixnum s;
        fixnum::add(s, a, b);
        r = s;
    }
};


template< typename fixnum >
struct add2 {
    __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
        fixnum s;
        fixnum::add2(s, a, b);
        r = s;
    }
};

template< typename fixnum >
struct bitwise_and {
    __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
        fixnum s;
        fixnum::bitwise_and(s, a, b);
        r = s;
    }
};

template< typename fixnum >
struct bitwise_xor {
    __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
        fixnum s;
        fixnum::bitwise_xor(s, a, b);
        r = s;
    }
};

template< typename fixnum >
struct bitwise_or {
    __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
        fixnum s;
        fixnum::bitwise_or(s, a, b);
        r = s;
    }
};



template< typename fixnum >
struct reverse_bits {
    __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
        fixnum s;
        fixnum::reverse_bits(s, a);
        r = s;
    }
};

template< typename fixnum >
struct mul_lo {
    __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
        fixnum s;
        fixnum::mul_lo(s, a, b);
        r = s;
    }
};

template< typename fixnum >
struct mul_wide {
    __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
        fixnum rr, ss;
        fixnum::mul_wide(ss, rr, a, b);
        r = ss;
    }
};

template< typename fixnum >
struct sqr_wide {
    __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
        fixnum rr, ss;
        fixnum::sqr_wide(ss, rr, a);
        r = ss;
    }
};

template< typename fixnum >
struct sqr_lo1 {
    __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
        fixnum s;
        fixnum::mul_lo(s, a, a);
        r = s;
    }
};

template< typename fixnum >
struct sqr_lo2 {
    __device__ void operator()(fixnum &r, fixnum a, fixnum b) {
        fixnum s;
        fixnum::sqr_lo(s, a);
        r = s;
    }
};

template< typename modnum >
struct my_modexp {
    typedef typename modnum::fixnum fixnum;

    __device__ void operator()(fixnum &z, fixnum x, fixnum y) {
        modexp<modnum> me(x, x);
        fixnum zz;
        me(zz, x);
        z = zz;
    };
};

template< typename modnum >
struct my_multi_modexp {
    typedef typename modnum::fixnum fixnum;

    __device__ void operator()(fixnum &z, fixnum x, fixnum y) {
        multi_modexp<modnum> mme(x);
        fixnum zz;
        mme(zz, x, x);
        z = zz;
    };
};

typedef warp_fixnum<32, u64_fixnum> fixnum_256;
typedef fixnum_array<fixnum_256> fixnum_array_256;

template< int fn_bytes, typename word_fixnum, template <typename> class Func >
void bench(int nelts) {
    typedef warp_fixnum<fn_bytes, word_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;

    if (nelts == 0) {
        puts(" -*-  nelts == 0; skipping...  -*-");
        return;
    }

    uint8_t *input1 = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i)
        input1[i] = (i * 17 + 11) % 256;

    uint8_t *input2 = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i)
        input2[i] = ((i + 13) * 17 + 11) % 256;

    fixnum_array *res, *in_a, *in_b;
    in_a = fixnum_array::create(input1, fn_bytes * nelts, fn_bytes);
    in_b = fixnum_array::create(input2, fn_bytes * nelts, fn_bytes);
    res = fixnum_array::create(nelts);

    // warm up
    fixnum_array::template map<Func>(res, in_a, in_b);

    clock_t c = clock();
    fixnum_array::template map<Func>(res, in_a, in_b);
    c = clock() - c;

    double secinv = (double)CLOCKS_PER_SEC / c;
    double total_MiB = fixnum::BYTES * (double)nelts / (1 << 20);
    printf(" %4d   %3d    %6.1f   %7.3f  %12.1f\n",
           fixnum::BITS, fixnum::digit::BITS, total_MiB,
           1/secinv, nelts * 1e-3 * secinv);

    delete in_a;
    delete in_b;
    delete res;
    delete[] input1;
    delete[] input2;
}

template< template <typename> class Func >
void bench_func(const char *fn_name, int nelts) {
    printf("Function: %s, #elts: %de3\n", fn_name, (int)(nelts * 1e-3));
    printf("fixnum digit  total data   time       Kops/s\n");
    printf(" bits  bits     (MiB)    (seconds)\n");
    // bench<4, u32_fixnum, Func>(nelts);
    bench<8, u32_fixnum, Func>(nelts);
    bench<16, u32_fixnum, Func>(nelts);
    bench<32, u32_fixnum, Func>(nelts);
    bench<64, u32_fixnum, Func>(nelts);
    // bench<128, u32_fixnum, Func>(nelts);
    puts("");

    bench<8, u64_fixnum, Func>(nelts);
    bench<16, u64_fixnum, Func>(nelts);
    bench<32, u64_fixnum, Func>(nelts);
    bench<64, u64_fixnum, Func>(nelts);
    // bench<128, u64_fixnum, Func>(nelts);
    // bench<256, u64_fixnum, Func>(nelts);
    puts("");
}

// typedef std::array<uint256_t, 10> digit_cache_t;

// template<int DEC_LENGTH>
// constexpr std::array<std::array<uint256_t, 10>, DEC_LENGTH> get_digit_cache() {
//     digit_cache_t ret;


//     return ret;
// }

// template<int N>
// struct A {
//     constexpr A() : arr() {
//         for (auto i = 0; i < N; ++i) {
//             for (auto i = 0; i < 10; ++i) {
//                 arr[i] = i;
//             }
//         }
//     }
//     std::array<std::array<int, 10>, N> arr;
// };



constexpr uint256_t get_for_digit(int pos, int digit, int dec_length) {
    uint256_t ret = uint256_t(10).pow(pos) * digit;
    int reverse_pos = dec_length - pos - 1;
    if (pos != reverse_pos) {
        ret += uint256_t(10).pow(reverse_pos) * digit;
    }

    return ret;
}

constexpr uint256_t get_max(int pos, int base, int length) {
    return uint256_t(base).pow(length - pos - 1) - uint256_t(base).pow(pos + 1);
}


template<int BASE, int DEC_LENGTH>
struct MaxCache {
    constexpr MaxCache() : arr() {
        for (auto i = 0; i < DEC_LENGTH; ++i) {
            arr[i] = get_max(i, BASE, DEC_LENGTH);
        }
    }
    uint256_t arr[DEC_LENGTH];
};



struct state_t {
    uint256_t current_num;
    uint256_t bin_num;
    int level;
};


template<int DEC_LENGTH>
void find_palindrome_recursive(state_t initial) {
    state_t stack[DEC_LENGTH * 10];
    stack[0] = initial;
    int stack_index = 0;

    while (stack_index >= 0) {
        uint256_t current_num = stack[stack_index].current_num;
        uint256_t bin_num = stack[stack_index].bin_num;
        uint256_t level = stack[stack_index].level;
        stack_index -= 1;

        if (level + 1 * 2 >= DEC_LENGTH) {
        }
    }
}



int main(int, char **) {
    // long m = 1;
    // if (argc > 1)
    //     m = atol(argv[1]);
    // m = std::max(m, 100000L);

    // bench_func<add>("add", m);
    // puts("");

    // bench_func<add2>("add2", m);
    // puts("");

    // uint256_t bla;
    // state_t initial = {
    //     uint256_t(0), uint256_t(0), 0
    // };
    // find_palindrome_recursive<10>(initial);

    constexpr auto max_cache_2 = MaxCache<2, 5>();
    // constexpr uint256_t bla = get_max(5, 5, 5);

    uint256_t d = get_for_digit(3, 3, 12);
    std::cout << d.str() << std::endl;

    uint256_t m = get_max(1, 10, 5);
    std::cout << m.str() << std::endl;


    return 0;
}
